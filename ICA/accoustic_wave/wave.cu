
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define DT 0.0070710676f // delta t
#define DX 15.0f // delta x
#define DY 15.0f // delta y
#define V 1500.0f // wave velocity v = 1500 m/s
#define HALF_LENGTH 1 // radius of the stencil

#define NUM_THREADS_BLOCK_X 32
#define NUM_THREADS_BLOCK_Y 32
 

__constant__ float dxSquaredGPU;
__constant__ float dySquaredGPU;
__constant__ float dtSquaredGPU;
__constant__ int rowsGPU;
__constant__ int colsGPU;

//--------------------------------------------------------------------------------------
// https://stackoverflow.com/questions/19646256/cudamemcpytosymbol-use-details
#define CUDA_CHECK_RETURN(value) {                                      \
    hipError_t _m_cudaStat = value;                                    \
    if (_m_cudaStat != hipSuccess) {                                   \
        fprintf(stderr, "Error %s at line %d in file %s\n",             \
                hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
    exit(1);                                                            \
} }
//--------------------------------------------------------------------------------------
/*
 * save the matrix on a file.txt
 */

void save_grid(int rows, int cols, float *matrix){
    system("mkdir -p wavefield");

    char file_name[64];
    sprintf(file_name, "wavefield/wavefield.txt");

    // save the result
    FILE *file;
    file = fopen(file_name, "w");

    for(int i = 0; i < rows; i++) {

        int offset = i * cols;

        for(int j = 0; j < cols; j++) {
            fprintf(file, "%f ", matrix[offset + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
    
    system("python3 plot.py");
}
//--------------------------------------------------------------------------------------
__global__ void wavekernel(float *prev_baseGPU,float * vel_baseGPU,float *next_baseGPU)
{    
    int  c = blockIdx.x * blockDim.x + threadIdx.x + HALF_LENGTH;
    int  r = blockIdx.y * blockDim.y + threadIdx.y + HALF_LENGTH;
    if (c < colsGPU - HALF_LENGTH && r < rowsGPU - HALF_LENGTH) {
        int idx = r * colsGPU + c;
        int doisPrevBaseIdx = 2.0 * prev_baseGPU[idx];
        float value = (prev_baseGPU[idx + 1] - doisPrevBaseIdx + prev_baseGPU[idx - 1]) / dxSquaredGPU;
        value += (prev_baseGPU[idx + colsGPU] - doisPrevBaseIdx + prev_baseGPU[idx - colsGPU]) / dySquaredGPU;      
        value *= dtSquaredGPU * vel_baseGPU[idx];      
        next_baseGPU[idx] = doisPrevBaseIdx - next_baseGPU[idx] + value;
    }
   
}

int main(int argc, char* argv[]) {

    if(argc != 4){
        printf("Usage: ./stencil N1 N2 TIME\n");
        printf("N1 N2: grid sizes for the stencil\n");
        printf("TIME: propagation time in ms\n");
        exit(-1);
    }
    
        hipSetDevice(0);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,0);

        system("echo 'Modelo Processador: '|cat /proc/cpuinfo|grep 'model name'|head -1");
        printf("Modelo do Device: %s\n",prop.name);
        printf("Número de SMs: %d\n",prop.multiProcessorCount);
        printf("Número de Regs por SM: %d K\n",prop.regsPerMultiprocessor >> 10);
        printf("Número de Regs por Bloco: %d K\n",prop.regsPerBlock  >> 10);
        printf("Memória compartilhada por SM: %lu KB\n",prop.sharedMemPerMultiprocessor >> 10);
        printf("Memória compartilhada por Bloco: %lu KB\n",prop.sharedMemPerBlock  >> 10);
        printf("Memória Global: %lu GB\n",prop.totalGlobalMem  >> 10  >> 10  >> 10 );
        printf("Memória Constante: %lu KB\n",prop.totalConstMem  >> 10);
    
    // number of rows of the grid
    int rows = atoi(argv[1]);

    // number of columns of the grid
    int cols = atoi(argv[2]);

    // number of timesteps
    int time = atoi(argv[3]);
    
    // calc the number of iterations (timesteps)
    int iterations = (int)((time/1000.0) / DT);

    // represent the matrix of wavefield as an array
    float *prev_base = (float*) malloc(rows * cols * sizeof(float));
    float *next_base = (float*) malloc(rows * cols * sizeof(float));

    // represent the matrix of velocities as an array
    float *vel_base =(float*) malloc(rows * cols * sizeof(float));
    // ************* BEGIN INITIALIZATION *************
    // define source wavelet
    float wavelet[12] = {0.016387336, -0.041464937, -0.067372555, 0.386110067,
                         0.812723635, 0.416998396,  0.076488599,  -0.059434419,
                         0.023680172, 0.005611435,  0.001823209,  -0.000720549};
    // initialize matrix
    for(int i = 0; i < rows; i++){

        int offset = i * cols;

        for(int j = 0; j < cols; j++){
            prev_base[offset + j] = 0.0f;
            next_base[offset + j] = 0.0f;
            vel_base[offset + j] = V * V;
        }
    }

    // add a source to initial wavefield as an initial condition
    for(int s = 11; s >= 0; s--){
        for(int i = rows / 2 - s; i < rows / 2 + s; i++){
            int offset = i * cols;
            for(int j = cols / 2 - s; j < cols / 2 + s; j++)
                prev_base[offset + j] = wavelet[s];
        }
    }
    // ************** END INITIALIZATION **************
    printf("Computing wavefield ... \n");
    // wavefield modeling
    float dxSquared = DX * DX;
    float dySquared = DY * DY;
    float dtSquared = DT * DT;

    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dxSquaredGPU), &dxSquared, sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dySquaredGPU), &dySquared, sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(dtSquaredGPU), &dtSquared, sizeof(float)));  
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(rowsGPU), &rows,sizeof(int)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(colsGPU), &cols,sizeof(int)));

    //--------------------------------------------------------------------------------------
    float *prev_baseGPU;
    float *next_baseGPU;
    float *vel_baseGPU;
    //--------------------------------------------------------------------------------------
    hipMalloc( (void**) &prev_baseGPU,rows * cols *sizeof(float));
    hipMalloc( (void**) &next_baseGPU,rows * cols *sizeof(float));
    hipMalloc( (void**) &vel_baseGPU, rows * cols *sizeof(float));
    //--------------------------------------------------------------------------------------
    hipMemcpy(prev_baseGPU, prev_base,rows * cols *sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(next_baseGPU, next_base,rows * cols *sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vel_baseGPU,  vel_base, rows * cols *sizeof(float), hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------------------
    dim3 bloco = dim3(NUM_THREADS_BLOCK_X, NUM_THREADS_BLOCK_Y);
    dim3 grid = dim3(ceil (rows/ (float)NUM_THREADS_BLOCK_X), ceil (cols/(float) NUM_THREADS_BLOCK_Y));
    //--------------------------------------------------------------------------------------
    float *swapGPU;
    //--------------------------------------------------------------------------------------
    hipEvent_t start, stop;
    float gpu_time = 0.0f;
        CUDA_CHECK_RETURN( hipEventCreate(&start) );
        CUDA_CHECK_RETURN( hipEventCreate(&stop) );
        CUDA_CHECK_RETURN( hipEventRecord(start, 0) );
    // launch kernal
    for(int n = 0; n < iterations; n++) {
        wavekernel<<<grid,bloco>>>(prev_baseGPU,vel_baseGPU,next_baseGPU);
        swapGPU = next_baseGPU;
        next_baseGPU = prev_baseGPU;
        prev_baseGPU = swapGPU;
    }

    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

    hipMemcpy(next_base, next_baseGPU, rows * cols *sizeof(float), hipMemcpyDeviceToHost);

    //Obtém o erro de lançamento de kernel
    CUDA_CHECK_RETURN( hipEventRecord(stop, 0) );
    CUDA_CHECK_RETURN( hipEventSynchronize(stop) );
    CUDA_CHECK_RETURN( hipEventElapsedTime(&gpu_time, start, stop) );

    printf("\nTempo de Execução na GPU: %f ms ", gpu_time);
    //--------------------------------------------------------------------------------------
    // limpando o que não vai ser mais usado
    hipFree(prev_baseGPU);hipFree(next_baseGPU);hipFree(vel_baseGPU);
    //--------------------------------------------------------------------------------------

    save_grid(rows, cols, next_base);


    free(prev_base);
    free(next_base);
    free(vel_base);

    return 0;
}

