
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
 
#define QTD_ELEMENTOS 2048
#define NUM_THREADS_BLOCK_X 32
#define NUM_THREADS_BLOCK_Y 32
 
void inicializaMatriz(int *data, unsigned size)
{
  time_t t;
    srand((unsigned int) time(&t));
    for (int i=0; i<size; i++) {
     for (int j=0; j<size; j++) {
             data[i * size + j] = ((int)rand() ) % 2;
        }
    }
}
 
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
 
 
void warshallCPU(int* fechoMatriz, unsigned n)
{
    for(int k = 0; k < n; k++){
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n; j++){
                    if(fechoMatriz[k * n + j] == 1 && fechoMatriz[i * n + k] == 1)  
                        fechoMatriz[i * n + j] = 1;
            }           
        }                   
    }
}
void imprimeSoma(int* data, unsigned n)
{
    double soma = 0;
    for (int i=0; i < n; i++) {
        for (int j=0; j < n; j++){
            soma += data[i * n + j];        
        }        
    } 
    printf("A soma é %f\n",soma);
}
 
__global__ void warshallGPU(int *A, int k, unsigned n)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int l = blockIdx.y * blockDim.y + threadIdx.y;
    // não tenho numero da interação.
    if(A[k * n + c] == 1 && A[l * n + k] == 1)
        A[l * n + c] = 1;      
 
}
 
void processamentoGPU(int *A ,unsigned n){
    //Aloca espaço na CPU para o resultado
    int matrizSize = sizeof(int) * n * n;
    // alocando o tamanho da matriz
    int* F = (int*) malloc(matrizSize);
    // ponteiros para gpu
    int* gA;
    hipMalloc( (void**) &gA, matrizSize);
    //-------------------------------------
    hipMemcpy(gA, A, matrizSize, hipMemcpyHostToDevice);
 
    dim3 bloco = dim3(NUM_THREADS_BLOCK_X, NUM_THREADS_BLOCK_Y);
    dim3 grid = dim3(ceil (n/ (float) NUM_THREADS_BLOCK_X), ceil (n/ (float) NUM_THREADS_BLOCK_Y));
 
    hipEvent_t start, stop;
    float gpu_time = 0.0f;
        checkCuda( hipEventCreate(&start) );
        checkCuda( hipEventCreate(&stop) );
        checkCuda( hipEventRecord(start, 0) );
 
    for (int k =0; k<n;k++){
        warshallGPU<<<grid,bloco>>>(gA,k, n);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        checkCuda( error );
    }
    
    //Obtém o erro de lançamento de kernel
    checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&gpu_time, start, stop) );
    //-------------------------------------------------------------
    hipMemcpy(A, gA, matrizSize, hipMemcpyDeviceToHost);
    hipFree(gA);
    //-------------------------------------------------------------
    //Imprime o resultado
    imprimeSoma(A, n);
    free(A);
        printf("Tempo de Execução na GPU: %.4f ms ", gpu_time);
}
 
void processamentoCPU(int *A, unsigned n)
{
    int* F = (int*) malloc( sizeof(int) * n * n);
    memcpy(F, A, sizeof(int)*n*n);
    double tempoGasto;
    
    clock_t start = clock();    
        warshallCPU(F, n);
    clock_t stop = clock();
    tempoGasto = (stop - start) / (float) CLOCKS_PER_SEC;
    printf("Tempo de execução da CPU: %f s\n", tempoGasto ); 
    imprimeSoma(F, n);
    free(F);
}
 
void mainWarshall()
{
 
    int byteNumber = QTD_ELEMENTOS * QTD_ELEMENTOS * sizeof(int);
 
    int *A = (int*) malloc(byteNumber);
 
    inicializaMatriz(A, QTD_ELEMENTOS);
    
    processamentoCPU(A, QTD_ELEMENTOS);
    //processamentoGPU(A, QTD_ELEMENTOS);
  
    free(A);
}
 
int main(void)
{
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);

    printf("Modelo Processador%s",system("cat /proc/cpuinfo|grep 'model name'|head -1"));
    printf("Modelo do Device: %s\n",prop.name);
    printf("Número de SMs: %d\n",prop.multiProcessorCount);
    printf("Número de Regs por SM: %d K\n",prop.regsPerMultiprocessor >> 10);
    printf("Número de Regs por Bloco: %d K\n",prop.regsPerBlock  >> 10);
    printf("Memória compartilhada por SM: %d KB\n",prop.sharedMemPerMultiprocessor >> 10);
    printf("Memória compartilhada por Bloco: %d KB\n",prop.sharedMemPerBlock  >> 10);
    printf("Memória Global: %d GB\n",prop.totalGlobalMem  >> 10  >> 10  >> 10 );
    printf("Memória Constante: %d KB\n",prop.totalConstMem  >> 10);
 
    mainWarshall();
    return 0;
}