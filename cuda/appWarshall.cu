
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>

#define QTD_ELEMENTOS 1024
#define NUM_THREADS_BLOCK_X 32
#define NUM_THREADS_BLOCK_Y 32

void inicializaMatriz(int *data, unsigned size)
{
  time_t t;
	srand((unsigned int) time(&t));
	for (int i=0; i<size; i++) {
     for (int j=0; j<size; j++) {
		   //data[i * size + j] = (int)( rand() & 0xFF )/10.0f;
			 data[i * size + j] = ((int)rand() ) % 2;
     	}
	}
}


void warshallCPU(int* A, int* F, unsigned n)
{
		for(int k = 0; k < n; k++){
			for(int lin = 0; lin < n; lin ++){
				for(int col = 0; col < n; col ++){
						if(A[k * n + col] == 1 && A[lin * n + k] == 1)
							F[lin * n + col] = 1;
				}
			}
		}
}
__global__ void warshallGPU(int k,int *A,int *F, unsigned n, int *R)
{
  
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  int l = blockIdx.y * blockDim.y + threadIdx.y;

  int i = c * n + l;
  __shared__ int best;
	if(threadIdx.x==0)
		best=A[n*blockIdx.y+k];
	__syncthreads();
  // não tenho numero da interação.
  if(A[k * n + c] == 1 && A[l * n + k] == 1)
		F[l * n + c] = 1;
  R[i] = A[i] + F[i];
	

}
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void imprimeSoma(int* data, unsigned n)
{
    double soma = 0;
    for (int i=0; i < n; i++) {
                for (int j=0; j < n; j++){
                    soma += data[i * n + j];        
                }        
      } 
    printf("A soma é %f\n",soma);
}
void processamentoGPU(int *A,int *B ,unsigned n){

	//Aloca espaço na CPU para o resultado
	int matrizSize = sizeof(int) * n * n;
	// alocando o tamanho da matriz
	int* F = (int*) malloc(matrizSize);

	// ponteiros para gpu

	int* gA;
	int* gB;
	int* gR;
	int* r=0;

	hipMalloc( (void**) &gA, matrizSize);
  	hipMalloc( (void**) &gB, matrizSize);
  	hipMalloc( (void**) &gR, matrizSize);

	//-------------------------------------

	hipMemcpy(gA, A, matrizSize, hipMemcpyHostToDevice);
    hipMemcpy(gB, B, matrizSize, hipMemcpyHostToDevice);

  	dim3 bloco = dim3(NUM_THREADS_BLOCK_X, NUM_THREADS_BLOCK_Y);
  	dim3 grid = dim3(ceil (n/ (float) NUM_THREADS_BLOCK_X), ceil (n/ (float) NUM_THREADS_BLOCK_Y));

	hipEvent_t start, stop;
    float gpu_time = 0.0f;
    	checkCuda( hipEventCreate(&start) );
    	checkCuda( hipEventCreate(&stop) );
    	checkCuda( hipEventRecord(start, 0) );

	for (int k =0; k<n;k++){
		warshallGPU<<<grid,bloco>>>(k,gA, gB, n,gR);
	}
	hipDeviceSynchronize();
	//Obtém o erro de lançamento de kernel
    hipError_t error = hipGetLastError();
    checkCuda( error );

 	checkCuda( hipEventRecord(stop, 0) );
    checkCuda( hipEventSynchronize(stop) );
    checkCuda( hipEventElapsedTime(&gpu_time, start, stop) );

	hipMemcpy(r, gR, matrizSize, hipMemcpyDeviceToHost);

	hipFree(gA);
  	hipFree(gB);
  	hipFree(gR);

  	//Imprime o resultado
  	imprimeSoma(r, n);
   		printf("Tempo de Execução na GPU: %.4f ms ", gpu_time);



}
void processamentoCPU(int *A, unsigned n)
{
	int* F = (int*) malloc( sizeof(int) * n * n);
  	double tempoGasto;
	clock_t start = clock();
	warshallCPU(A, F, n);
	clock_t stop = clock();
	tempoGasto = 1000 *  (stop - start) / (float) CLOCKS_PER_SEC;
	printf("Tempo de execução da CPU: %f ms\n", tempoGasto );
	free(F);
}

void mainWarshall()
{

	int byteNumber = QTD_ELEMENTOS * QTD_ELEMENTOS * sizeof(int);

	int *A = (int*) malloc(byteNumber);

    int *B = (int*) malloc(byteNumber);
	inicializaMatriz(A, QTD_ELEMENTOS);
	inicializaMatriz(B, QTD_ELEMENTOS);

	
	processamentoCPU(A, QTD_ELEMENTOS);
	processamentoGPU(A,B, QTD_ELEMENTOS);
  
	free(A);
	free(B);
}

int main(void)
{

	hipSetDevice(0);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	printf("Número de SM: %d\n",prop.multiProcessorCount);
	printf("Modelo GPU: %s\n",prop.name);

	mainWarshall();
	return 0;
}
